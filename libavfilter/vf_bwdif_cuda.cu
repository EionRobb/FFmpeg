
#include <hip/hip_runtime.h>
/*
 * Copyright (C) 2018 Philip Langdale <philipl@overt.org>
 *
 * This file is part of FFmpeg.
 *
 * FFmpeg is free software; you can redistribute it and/or
 * modify it under the terms of the GNU Lesser General Public
 * License as published by the Free Software Foundation; either
 * version 2.1 of the License, or (at your option) any later version.
 *
 * FFmpeg is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU
 * Lesser General Public License for more details.
 *
 * You should have received a copy of the GNU Lesser General Public
 * License along with FFmpeg; if not, write to the Free Software
 * Foundation, Inc., 51 Franklin Street, Fifth Floor, Boston, MA 02110-1301 USA
 */

__device__ static const unsigned short coef_lf[2] = { 4309, 213 };
__device__ static const unsigned short coef_hf[3] = { 5570, 3801, 1016 };
__device__ static const unsigned short coef_sp[2] = { 5077, 981 };

template<typename T>
__inline__ __device__ T max3(T a, T b, T c)
{
    T x = max(a, b);
    return max(x, c);
}

template<typename T>
__inline__ __device__ T min3(T a, T b, T c)
{
    T x = min(a, b);
    return min(x, c);
}

template<typename T>
__inline__ __device__ T clip(T a, T min, T max)
{
    if (a < min) {
        return min;
    } else if (a > max) {
        return max;
    } else {
        return a;
    }
}

template<typename T>
__inline__ __device__ T filter(T A, T B, T C, T D,
                               T a, T b, T c, T d, T e, T f, T g,
                               T h, T i, T j, T k, T l, T m, T n,
                               int clip_max)
{
    T final;

    int fc = C;
    int fd = (c + l) >> 1;
    int fe = B;

    int temporal_diff0 = abs(c - l);
    int temporal_diff1 = (abs(g - fc) + abs(f - fe)) >> 1;
    int temporal_diff2 = (abs(i - fc) + abs(h - fe)) >> 1;
    int diff = max3(temporal_diff0 >> 1, temporal_diff1, temporal_diff2);

    if (!diff) {
        final = fd;
    } else {
        int fb = ((d + m) >> 1) - fc;
        int ff = ((c + l) >> 1) - fe;
        int dc = fd - fc;
        int de = fd - fe;
        int mmax = max3(de, dc, min(fb, ff));
        int mmin = min3(de, dc, max(fb, ff));
        diff = max3(diff, mmin, -mmax);

        int interpol;
        if (abs(fc - fe) > temporal_diff0) {
            interpol = (((coef_hf[0] * (c + l)
                - coef_hf[1] * (d + m + b + k)
                + coef_hf[2] * (e + n + a + j)) >> 2)
                + coef_lf[0] * (C + B) - coef_lf[1] * (D + A)) >> 13;
        } else {
            interpol = (coef_sp[0] * (C + B) - coef_sp[1] * (D + A)) >> 13;
        }
        if (interpol > fd + diff) {
            interpol = fd + diff;
        } else if (interpol < fd - diff) {
            interpol = fd - diff;
        }
        final = clip(interpol, 0, clip_max);
    }

    return final;
}

template<typename T>
__inline__ __device__ void bwdif_single(T *dst,
                                        hipTextureObject_t prev,
                                        hipTextureObject_t cur,
                                        hipTextureObject_t next,
                                        int dst_width, int dst_height, int dst_pitch,
                                        int src_width, int src_height,
                                        int parity, int tff, bool skip_spatial_check,
                                        int clip_max)
{
    // Identify location
    int xo = blockIdx.x * blockDim.x + threadIdx.x;
    int yo = blockIdx.y * blockDim.y + threadIdx.y;

    if (xo >= dst_width || yo >= dst_height) {
        return;
    }

    // Don't modify the primary field
    if (yo % 2 == parity) {
      dst[yo*dst_pitch+xo] = tex2D<T>(cur, xo, yo);
      return;
    }

    T A = tex2D<T>(cur, xo, yo + 3);
    T B = tex2D<T>(cur, xo, yo + 1);
    T C = tex2D<T>(cur, xo, yo - 1);
    T D = tex2D<T>(cur, xo, yo - 3);

    // Calculate temporal prediction
    int is_second_field = !(parity ^ tff);

    hipTextureObject_t prev2 = prev;
    hipTextureObject_t prev1 = is_second_field ? cur : prev;
    hipTextureObject_t next1 = is_second_field ? next : cur;
    hipTextureObject_t next2 = next;

    T a = tex2D<T>(prev2, xo,  yo + 4);
    T b = tex2D<T>(prev2, xo,  yo + 2);
    T c = tex2D<T>(prev2, xo,  yo + 0);
    T d = tex2D<T>(prev2, xo,  yo - 2);
    T e = tex2D<T>(prev2, xo,  yo - 4);
    T f = tex2D<T>(prev1, xo,  yo + 1);
    T g = tex2D<T>(prev1, xo,  yo - 1);
    T h = tex2D<T>(next1, xo,  yo + 1);
    T i = tex2D<T>(next1, xo,  yo - 1);
    T j = tex2D<T>(next2, xo,  yo + 4);
    T k = tex2D<T>(next2, xo,  yo + 2);
    T l = tex2D<T>(next2, xo,  yo + 0);
    T m = tex2D<T>(next2, xo,  yo - 2);
    T n = tex2D<T>(next2, xo,  yo - 4);

    dst[yo*dst_pitch+xo] = filter(A, B, C, D,
                                  a, b, c, d, e, f, g,
                                  h, i, j, k, l, m, n,
                                  clip_max);
}

template <typename T>
__inline__ __device__ void bwdif_double(T *dst,
                                        hipTextureObject_t prev,
                                        hipTextureObject_t cur,
                                        hipTextureObject_t next,
                                        int dst_width, int dst_height, int dst_pitch,
                                        int src_width, int src_height,
                                        int parity, int tff, bool skip_spatial_check,
                                        int clip_max)
{
    int xo = blockIdx.x * blockDim.x + threadIdx.x;
    int yo = blockIdx.y * blockDim.y + threadIdx.y;

    if (xo >= dst_width || yo >= dst_height) {
        return;
    }

    if (yo % 2 == parity) {
      // Don't modify the primary field
      dst[yo*dst_pitch+xo] = tex2D<T>(cur, xo, yo);
      return;
    }

    T A = tex2D<T>(cur, xo, yo + 3);
    T B = tex2D<T>(cur, xo, yo + 1);
    T C = tex2D<T>(cur, xo, yo - 1);
    T D = tex2D<T>(cur, xo, yo - 3);

    // Calculate temporal prediction
    int is_second_field = !(parity ^ tff);

    hipTextureObject_t prev2 = prev;
    hipTextureObject_t prev1 = is_second_field ? cur : prev;
    hipTextureObject_t next1 = is_second_field ? next : cur;
    hipTextureObject_t next2 = next;

    T a = tex2D<T>(prev2, xo,  yo + 4);
    T b = tex2D<T>(prev2, xo,  yo + 2);
    T c = tex2D<T>(prev2, xo,  yo + 0);
    T d = tex2D<T>(prev2, xo,  yo - 2);
    T e = tex2D<T>(prev2, xo,  yo - 4);
    T f = tex2D<T>(prev1, xo,  yo + 1);
    T g = tex2D<T>(prev1, xo,  yo - 1);
    T h = tex2D<T>(next1, xo,  yo + 1);
    T i = tex2D<T>(next1, xo,  yo - 1);
    T j = tex2D<T>(next2, xo,  yo + 4);
    T k = tex2D<T>(next2, xo,  yo + 2);
    T l = tex2D<T>(next2, xo,  yo + 0);
    T m = tex2D<T>(next2, xo,  yo - 2);
    T n = tex2D<T>(next2, xo,  yo - 4);

    T final;
    final.x = filter(A.x, B.x, C.x, D.x,
                     a.x, b.x, c.x, d.x, e.x, f.x, g.x,
                     h.x, i.x, j.x, k.x, l.x, m.x, n.x,
                     clip_max);
    final.y = filter(A.y, B.y, C.y, D.y,
                     a.y, b.y, c.y, d.y, e.y, f.y, g.y,
                     h.y, i.y, j.y, k.y, l.y, m.y, n.y,
                     clip_max);




    dst[yo*dst_pitch+xo] = final;
}

extern "C" {

__global__ void bwdif_uchar(unsigned char *dst,
                            hipTextureObject_t prev,
                            hipTextureObject_t cur,
                            hipTextureObject_t next,
                            int dst_width, int dst_height, int dst_pitch,
                            int src_width, int src_height,
                            int parity, int tff, bool skip_spatial_check,
                            int clip_max)
{
    bwdif_single(dst, prev, cur, next,
                 dst_width, dst_height, dst_pitch,
                 src_width, src_height,
                 parity, tff, skip_spatial_check,
                 clip_max);
}

__global__ void bwdif_ushort(unsigned short *dst,
                            hipTextureObject_t prev,
                            hipTextureObject_t cur,
                            hipTextureObject_t next,
                            int dst_width, int dst_height, int dst_pitch,
                            int src_width, int src_height,
                            int parity, int tff, bool skip_spatial_check,
                            int clip_max)
{
    bwdif_single(dst, prev, cur, next,
                 dst_width, dst_height, dst_pitch,
                 src_width, src_height,
                 parity, tff, skip_spatial_check,
                 clip_max);
}

__global__ void bwdif_uchar2(uchar2 *dst,
                            hipTextureObject_t prev,
                            hipTextureObject_t cur,
                            hipTextureObject_t next,
                            int dst_width, int dst_height, int dst_pitch,
                            int src_width, int src_height,
                            int parity, int tff, bool skip_spatial_check,
                            int clip_max)
{
    bwdif_double(dst, prev, cur, next,
                 dst_width, dst_height, dst_pitch,
                 src_width, src_height,
                 parity, tff, skip_spatial_check,
                 clip_max);
}

__global__ void bwdif_ushort2(ushort2 *dst,
                            hipTextureObject_t prev,
                            hipTextureObject_t cur,
                            hipTextureObject_t next,
                            int dst_width, int dst_height, int dst_pitch,
                            int src_width, int src_height,
                            int parity, int tff, bool skip_spatial_check,
                            int clip_max)
{
    bwdif_double(dst, prev, cur, next,
                 dst_width, dst_height, dst_pitch,
                 src_width, src_height,
                 parity, tff, skip_spatial_check,
                 clip_max);
}

} /* extern "C" */
